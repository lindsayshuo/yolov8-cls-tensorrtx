#include "hip/hip_runtime.h"
#include "yololayer.h"
#include "types.h"
#include <assert.h>
#include <math.h>
#include "cuda_utils.h"
#include <vector>
#include <iostream>

namespace Tn {
    template<typename T>
    void write(char*& buffer, const T& val) {
        *reinterpret_cast<T*>(buffer) = val;
        buffer += sizeof(T);
    }

    template<typename T>
    void read(const char*& buffer, T& val) {
        val = *reinterpret_cast<const T*>(buffer);
        buffer += sizeof(T);
    }
}  // namespace Tn


namespace nvinfer1 {
YoloLayerPlugin::YoloLayerPlugin(int classCount, int netWidth, int netHeight, int maxOut, bool is_segmentation) {
    mClassCount = classCount;
    mYoloV8NetWidth = netWidth;
    mYoloV8netHeight = netHeight;
    mMaxOutObject = maxOut;
    is_segmentation_ = is_segmentation;
}

YoloLayerPlugin::~YoloLayerPlugin() {}

YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length) {
    using namespace Tn;
    const char* d = reinterpret_cast<const char*>(data), * a = d;
    read(d, mClassCount);
    read(d, mThreadCount);
    read(d, mYoloV8NetWidth);
    read(d, mYoloV8netHeight);
    read(d, mMaxOutObject);
    read(d, is_segmentation_);

    assert(d == a + length);
}

void YoloLayerPlugin::serialize(void* buffer) const TRT_NOEXCEPT {

    using namespace Tn;
    char* d = static_cast<char*>(buffer), * a = d;
    write(d, mClassCount);
    write(d, mThreadCount);
    write(d, mYoloV8NetWidth);
    write(d, mYoloV8netHeight);
    write(d, mMaxOutObject);
    write(d, is_segmentation_);

    assert(d == a + getSerializationSize());
}

size_t YoloLayerPlugin::getSerializationSize() const TRT_NOEXCEPT {
    return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mYoloV8netHeight) + sizeof(mYoloV8NetWidth) + sizeof(mMaxOutObject) + sizeof(is_segmentation_);
}

int YoloLayerPlugin::initialize() TRT_NOEXCEPT {
    return 0;
}

nvinfer1::Dims YoloLayerPlugin::getOutputDimensions(int index, const nvinfer1::Dims* inputs, int nbInputDims) TRT_NOEXCEPT {
    int total_size = mMaxOutObject * sizeof(Detection) / sizeof(float);
    return nvinfer1::Dims3(total_size + 1, 1, 1);
}

void YoloLayerPlugin::setPluginNamespace(const char* pluginNamespace) TRT_NOEXCEPT {
    mPluginNamespace = pluginNamespace;
}

const char* YoloLayerPlugin::getPluginNamespace() const TRT_NOEXCEPT {
    return mPluginNamespace;
}

nvinfer1::DataType YoloLayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const TRT_NOEXCEPT {
    return nvinfer1::DataType::kFLOAT;
}

bool YoloLayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const TRT_NOEXCEPT {

    return false;
}

bool YoloLayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const TRT_NOEXCEPT {

    return false;
}

void YoloLayerPlugin::configurePlugin(nvinfer1::PluginTensorDesc const* in, int nbInput, nvinfer1::PluginTensorDesc const* out, int nbOutput) TRT_NOEXCEPT {};

void YoloLayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator) TRT_NOEXCEPT {};

void YoloLayerPlugin::detachFromContext() TRT_NOEXCEPT {}

const char* YoloLayerPlugin::getPluginType() const TRT_NOEXCEPT {

    return "YoloLayer_TRT";
}

const char* YoloLayerPlugin::getPluginVersion() const TRT_NOEXCEPT {
    return "1";
}

void YoloLayerPlugin::destroy() TRT_NOEXCEPT {

    delete this;
}

nvinfer1::IPluginV2IOExt* YoloLayerPlugin::clone() const TRT_NOEXCEPT {

    YoloLayerPlugin* p = new YoloLayerPlugin(mClassCount, mYoloV8NetWidth, mYoloV8netHeight, mMaxOutObject, is_segmentation_);
    p->setPluginNamespace(mPluginNamespace);
    return p;
}

int YoloLayerPlugin::enqueue(int batchSize, const void* TRT_CONST_ENQUEUE* inputs, void* const* outputs, void* workspace, hipStream_t stream) TRT_NOEXCEPT {

    forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, mYoloV8netHeight, mYoloV8NetWidth, batchSize);
    return 0;
}


__device__ float Logist(float data) { return 1.0f / (1.0f + expf(-data)); };

__global__ void CalDetection(const float* input, float* output, int numElements, int maxoutobject,
                             const int grid_h, int grid_w, const int stride, int classes, int outputElem, bool is_segmentation) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= numElements) return;

    int total_grid = grid_h * grid_w;
    int info_len = 4 + classes;
    if (is_segmentation) info_len += 32;
    int batchIdx = idx / total_grid;
    int elemIdx = idx % total_grid;
    const float* curInput = input + batchIdx * total_grid * info_len;
    int outputIdx = batchIdx * outputElem;

    int class_id = 0;
    float max_cls_prob = 0.0;
    for (int i = 4; i < 4 + classes; i++) {
        float p = Logist(curInput[elemIdx + i * total_grid]);
        if (p > max_cls_prob) {
            max_cls_prob = p;
            class_id = i - 4;
        }
    }

    if (max_cls_prob < 0.1) return;

    int count = (int)atomicAdd(output + outputIdx, 1);
    if (count >= maxoutobject) return;
    char* data = (char*)(output + outputIdx) + sizeof(float) + count * sizeof(Detection);
    Detection* det = (Detection*)(data);

    int row = elemIdx / grid_w;
    int col = elemIdx % grid_w;

    det->conf = max_cls_prob;
    det->class_id = class_id;
    det->bbox[0] = (col + 0.5f - curInput[elemIdx + 0 * total_grid]) * stride;
    det->bbox[1] = (row + 0.5f - curInput[elemIdx + 1 * total_grid]) * stride;
    det->bbox[2] = (col + 0.5f + curInput[elemIdx + 2 * total_grid]) * stride;
    det->bbox[3] = (row + 0.5f + curInput[elemIdx + 3 * total_grid]) * stride;

    for (int k = 0; is_segmentation && k < 32; k++) {
        det->mask[k] = curInput[elemIdx + (k + 4 + classes) * total_grid];
    }
}

void YoloLayerPlugin::forwardGpu(const float* const* inputs, float* output, hipStream_t stream, int mYoloV8netHeight,int mYoloV8NetWidth, int batchSize) {
    int outputElem = 1 + mMaxOutObject * sizeof(Detection) / sizeof(float);
    hipMemsetAsync(output, 0, sizeof(float), stream);
    for (int idx = 0; idx < batchSize; ++idx) {
        CUDA_CHECK(hipMemsetAsync(output + idx * outputElem, 0, sizeof(float), stream));
    }
    int numElem = 0;
    int grids[3][2] = { {mYoloV8netHeight / 8, mYoloV8NetWidth / 8}, {mYoloV8netHeight / 16, mYoloV8NetWidth / 16}, {mYoloV8netHeight / 32, mYoloV8NetWidth / 32} };
    int strides[] = { 8, 16, 32 };
    for (unsigned int i = 0; i < 3; i++) {
        int grid_h = grids[i][0];
        int grid_w = grids[i][1];
        int stride = strides[i];
        numElem = grid_h * grid_w * batchSize;
        if (numElem < mThreadCount) mThreadCount = numElem;

        CalDetection << <(numElem + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream >> >
            (inputs[i], output, numElem, mMaxOutObject, grid_h, grid_w, stride, mClassCount, outputElem, is_segmentation_);
    }
}

PluginFieldCollection YoloPluginCreator::mFC{};
std::vector<PluginField> YoloPluginCreator::mPluginAttributes;

YoloPluginCreator::YoloPluginCreator() {
    mPluginAttributes.clear();
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* YoloPluginCreator::getPluginName() const TRT_NOEXCEPT {
    return "YoloLayer_TRT";
}

const char* YoloPluginCreator::getPluginVersion() const TRT_NOEXCEPT {
    return "1";
}

const PluginFieldCollection* YoloPluginCreator::getFieldNames() TRT_NOEXCEPT {
    return &mFC;
}

IPluginV2IOExt* YoloPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) TRT_NOEXCEPT {
    assert(fc->nbFields == 1);
    assert(strcmp(fc->fields[0].name, "netinfo") == 0);
    int* p_netinfo = (int*)(fc->fields[0].data);
    int class_count = p_netinfo[0];
    int input_w = p_netinfo[1];
    int input_h = p_netinfo[2];
    int max_output_object_count = p_netinfo[3];
    bool is_segmentation = p_netinfo[4];
    YoloLayerPlugin* obj = new YoloLayerPlugin(class_count, input_w, input_h, max_output_object_count, is_segmentation);
    obj->setPluginNamespace(mNamespace.c_str());
    return obj;
}

IPluginV2IOExt* YoloPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength) TRT_NOEXCEPT {
    // This object will be deleted when the network is destroyed, which will
    // call YoloLayerPlugin::destroy()
    YoloLayerPlugin* obj = new YoloLayerPlugin(serialData, serialLength);
    obj->setPluginNamespace(mNamespace.c_str());
    return obj;
}

} // namespace nvinfer1
